#include "hip/hip_runtime.h"
/*******************************************************************************
 *  Name: Joshua Le Gresley 
 * 
 *  Purpose: This program uses Nvidia CUDA C to create a parallel 
 *  implementation of the Mandelbrot algorithm. It produces a bitmap image 
 *  which contains the fractal representation of the Mandelbrot set. Program
 *  variables can be set from the commandline and include resolution, width,
 *  height, xcenter and ycenter.
 *  The program utilises code sourced from UNE COSC330 lectures and practicals.
 *
 *  Desrciption: To compile use the provided makefile by running the 'make'
 *  command. To run the exectutable use the command in the format:
 *
 *     ./mandelbrot_parallel <resolution> <width> <height> <xcenter>
 *                                  <ycenter>"
 *
 *  resolution - is a double value for the 'zoom' level of the image.
 *  width - is an int value for the width of the image
 *  height - is an int value for the height of the image
 *  xcenter - is a double value for X Position of the fractal in the image 
 *             between 1 and -1
 *  ycenter - is a double value for Y Position of the fractal in the image 
 *             between 1 and -1
 ******************************************************************************/

#include "bmpfile.h"
#include <stdio.h>
/*Mandelbrot values*/
#define MAX_ITER 1000
/*Colour Values*/
#define COLOUR_DEPTH 255
#define COLOUR_MAX 240.0
#define GRADIENT_COLOUR_MAX 230.0

#define FILENAME "my_mandelbrot_fractal.bmp"

typedef struct {
        double resolution;
        int width;
        int height;
        int *x;
        int *y;
        double xcenter;
        double ycenter;
        int xoffset;
        int yoffset;
        rgb_pixel_t *image;
} Matrix;

/**
 * Computes the color gradiant
 * color: the output vector
 * x: the gradiant (beetween 0 and 360)
 * min and max: variation of the RGB channels (Move3D 0 -> 1)
 * Check wiki for more details on the colour science:
 * en.wikipedia.org/wiki/HSL_and_HSV
 */
__device__ void ground_color_mix(double *color, double x, double min,
                                 double max)
{
        /*
         * Red = 0
         * Green = 1
         * Blue = 2
         */
        double pos_slope = (max - min) / 60;
        double neg_slope = (min - max) / 60;

        if (x < 60) {
                color[0] = max;
                color[1] = pos_slope * x + min;
                color[2] = min;
                return;
        } else if (x < 120) {
                color[0] = neg_slope * x + 2.0 * max + min;
                color[1] = max;
                color[2] = min;
                return;
        } else if (x < 180) {
                color[0] = min;
                color[1] = max;
                color[2] = pos_slope * x - 2.0 * max + min;
                return;
        } else if (x < 240) {
                color[0] = min;
                color[1] = neg_slope * x + 4.0 * max + min;
                color[2] = max;
                return;
        } else if (x < 300) {
                color[0] = pos_slope * x - 4.0 * max + min;
                color[1] = min;
                color[2] = max;
                return;
        } else {
                color[0] = max;
                color[1] = min;
                color[2] = neg_slope * x + 6 * max;
                return;
        }
}

/**
 * CUDA Kernel Device code
 * Determines where the pixel isvreferencing in the mandelbrot set.
 * Then sets the pixel colour.
 */
__global__ void mandelbrot(Matrix mandel_grid)
{
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        rgb_pixel_t pixel = {0, 0, 0, 0};

        if (i < mandel_grid.width * mandel_grid.height) {
                /* Determine where in the mandelbrot set, the pixel is
                referencing */

                double x = mandel_grid.xcenter +
                           (mandel_grid.xoffset + mandel_grid.x[i]) /
                               mandel_grid.resolution;
                double y = mandel_grid.ycenter +
                           (mandel_grid.yoffset - mandel_grid.y[i]) /
                               mandel_grid.resolution;

                /* Mandelbrot stuff */
                double a = 0;
                double b = 0;
                double aold = 0;
                double bold = 0;
                double zmagsqr = 0;
                int iter = 0;
                double x_col;
                double color[3];

                /* Check if the x,y coord are part of the mendelbrot set
                - refer to the algorithm */
                while (iter < MAX_ITER && zmagsqr <= 4.0) {
                        ++iter;
                        a = (aold * aold) - (bold * bold) + x;
                        b = 2.0 * aold * bold + y;

                        zmagsqr = a * a + b * b;

                        aold = a;
                        bold = b;
                }

                /* Generate the colour of the pixel from the iter value */
                /* You can mess around with the colour settings to use
                 * different gradients */
                /* Colour currently maps from royal blue to red */
                x_col = (COLOUR_MAX - ((((float)iter / ((float)MAX_ITER) *
                                         GRADIENT_COLOUR_MAX))));
                ground_color_mix(color, x_col, 1, COLOUR_DEPTH);
                pixel.red = color[0];
                pixel.green = color[1];
                pixel.blue = color[2];
                mandel_grid.image[i] = pixel;
        }
}

/*
 * Validates the command line arguments.
 * parameters:
 *   argc - An int for the number of command line args passed in
 *   *argv -  A pointer to char array storing the passed in command line args
 *   *res - An int pointer for zoom of the image
 *   *width - An int pointer for width of image
 *   *height - An int pointer for height of image
 *   *xcenter - A double pointer for image center, +- 1.0
 *   *ycenter - A double pointer for image center, +- 1.0
 * returns: 0 if command line args are valid, -1 if they are invalid
 */
int parse_args(int argc, char *argv[], double *res, int *width, int *height,
               double *xcenter, double *ycenter)
{
        if ((argc != 6) || (*res = atof(argv[1])) <= 0.0 ||
            (*width = atoi(argv[2])) <= 0.0 ||
            (*height = atoi(argv[3])) <= 0.0 ||
            (*xcenter = atof(argv[4])) > 1.0 ||
            (*ycenter = atof(argv[5])) >= 1.0 ||
            (*xcenter = atof(argv[4])) < -1.0 ||
            (*ycenter = atof(argv[5])) <= -1.0) {
                fprintf(stderr,
                        "Usage: %s <resolution> "
                        "<width> <height> <xcenter> <ycenter>\n",
                        argv[0]);
                return (-1);
        }
        return 0;
}

/**
 * Host code
 */
int main(int argc, char **argv)
{
        int i = 0, col, row, num_elements;
        /* Error code to check return values for CUDA calls */
        hipError_t err = hipSuccess;
        Matrix mandel_grid;
        bmpfile_t *bmp;

        /* get input parameters */
        if (parse_args(argc, argv, &mandel_grid.resolution, &mandel_grid.width,
                       &mandel_grid.height, &mandel_grid.xcenter,
                       &mandel_grid.ycenter) < 0) {
                exit(EXIT_FAILURE);
        }

        /* Allocate the host memory */
        num_elements = mandel_grid.width * mandel_grid.height;
        size_t size = num_elements * sizeof(rgb_pixel_t);
        mandel_grid.image = (rgb_pixel_t *)malloc(size);

        bmp = bmp_create(mandel_grid.width, mandel_grid.height, 32);

        size_t grid_size = num_elements * sizeof(int);
        mandel_grid.x = (int *)malloc(grid_size);
        mandel_grid.y = (int *)malloc(grid_size);

        /* Verify that allocation succeeded */
        if (mandel_grid.image == NULL || bmp == NULL || mandel_grid.x == NULL 
                || mandel_grid.y == NULL) {
                fprintf(stderr, "Failed to allocate host memory!\n");
                exit(EXIT_FAILURE);
        }

        /* determine pixel grid position */
        for (col = 0; col < mandel_grid.width; col++) {
                for (row = 0; row < mandel_grid.height; row++) {
                        mandel_grid.x[col * mandel_grid.height + row] = col;
                        mandel_grid.y[col * mandel_grid.height + row] = row;
                }
        }

        /* calculate offset */
        mandel_grid.xoffset = -(mandel_grid.width - 1) / 2;
        mandel_grid.yoffset = (mandel_grid.height - 1) / 2;

        /* Now we will need to copy everything over to the GPU*/
        /* Allocate the device memory */
        Matrix d_mandel_grid;
        d_mandel_grid.resolution = mandel_grid.resolution;
        d_mandel_grid.width = mandel_grid.width;
        d_mandel_grid.height = mandel_grid.height;
        d_mandel_grid.xcenter = mandel_grid.xcenter;
        d_mandel_grid.ycenter = mandel_grid.ycenter;
        d_mandel_grid.xoffset = mandel_grid.xoffset;
        d_mandel_grid.yoffset = mandel_grid.yoffset;

        err = hipMalloc(&d_mandel_grid.image, size);
        if (err != hipSuccess) {
                fprintf(
                    stderr,
                    "Failed to allocate device image matrix "
                    "(error code %s)!\n",
                    hipGetErrorString(err));
                exit(EXIT_FAILURE);
        }

        err = hipMalloc(&d_mandel_grid.x, grid_size);
        if (err != hipSuccess) {
                fprintf(stderr,
                        "Failed to allocate device x coordinates "
                        "(error code %s)!\n",
                        hipGetErrorString(err));
                exit(EXIT_FAILURE);
        }

        err = hipMalloc(&d_mandel_grid.y, grid_size);
        if (err != hipSuccess) {
                fprintf(stderr,
                        "Failed to allocate device y coordinate matrix "
                        "(error code %s)!\n",
                        hipGetErrorString(err));
                exit(EXIT_FAILURE);
        }

        /* copy memory to cuda device */
        err = hipMemcpy(d_mandel_grid.image, mandel_grid.image, size,
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr,
                        "Failed to copy image from host to device "
                        "(error code %s)!\n",
                        hipGetErrorString(err));
                exit(EXIT_FAILURE);
        }

        err = hipMemcpy(d_mandel_grid.x, mandel_grid.x, grid_size,
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr,
                        "Failed to copy x coordinates from host to device "
                        "(error code %s)!\n",
                        hipGetErrorString(err));
                exit(EXIT_FAILURE);
        }

        err = hipMemcpy(d_mandel_grid.y, mandel_grid.y, grid_size,
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr,
                        "Failed to copy y coordinates from host to device "
                        "(error code %s)!\n",
                        hipGetErrorString(err));
                exit(EXIT_FAILURE);
        }

        /* Launch the mandelbrot CUDA Kernel */
        int threadsPerBlock = 256;
        int blocksPerGrid =
            (num_elements + threadsPerBlock - 1) / threadsPerBlock;
        printf("CUDA kernel launch with %d blocks of %d threads\n",
               blocksPerGrid, threadsPerBlock);

        mandelbrot<<<blocksPerGrid, threadsPerBlock>>>(d_mandel_grid);
        err = hipGetLastError();

        if (err != hipSuccess) {
                fprintf(stderr,
                        "Failed to launch mandelbrot kernel "
                        "(error code %s)!\n",
                        hipGetErrorString(err));
                exit(EXIT_FAILURE);
        }

        /* copy memory back from device */
        err = hipMemcpy(mandel_grid.image, d_mandel_grid.image, size,
                         hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
                fprintf(stderr,
                        "Failed to copy image from host to device "
                        "(error code %s)!\n",
                        hipGetErrorString(err));
                exit(EXIT_FAILURE);
        }

        /* set pixels from device calculations in bmp file */
        for (i = 0; i < num_elements; i++) {
                bmp_set_pixel(bmp, mandel_grid.x[i], mandel_grid.y[i],
                              mandel_grid.image[i]);
        }

        bmp_save(bmp, FILENAME);
        // Free device memory
        hipFree(d_mandel_grid.image);
        hipFree(d_mandel_grid.x);
        hipFree(d_mandel_grid.y);
        free(mandel_grid.image);
        free(mandel_grid.x);
        free(mandel_grid.y);
        bmp_destroy(bmp);

        return 0;
}
